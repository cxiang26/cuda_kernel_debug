#include "hip/hip_runtime.h"
#include <iostream>
#include "debug_utils.h"
using namespace std;

#define CUDA_KERNEL_LOOP(i, n) \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
         i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 512;

inline int32_t GET_BLOCKS(int32_t const N, int32_t const numThreads)
{
    return (N + numThreads - 1) / numThreads;
}


template <typename scalar_t>
__device__ scalar_t ms_deform_attn_im2col_bilinear(scalar_t const*& bottomData, int32_t const& height, int32_t const& width,
    int32_t const& nHeads, int32_t const& channels, scalar_t const& h, scalar_t const& w, int32_t const& m, int32_t const& c)
{
    int32_t const hLow = floor(h);
    int32_t const wLow = floor(w);
    int32_t const hHigh = hLow + 1;
    int32_t const wHigh = wLow + 1;

    scalar_t const lh = h - hLow;
    scalar_t const lw = w - wLow;
    scalar_t const hh = 1 - lh, hw = 1 - lw;

    int32_t const wStride = nHeads * channels;
    int32_t const hStride = width * wStride;
    int32_t const hLowPtrOffset = hLow * hStride;
    int32_t const hHighPtrOffset = hLowPtrOffset + hStride;
    int32_t const wLowPtrOffset = wLow * wStride;
    int32_t const wHighPtrOffset = wLowPtrOffset + wStride;
    int32_t const basePtr = m * channels + c;

    scalar_t v1 = 0;
    if (hLow >= 0 && wLow >= 0)
    {
        int32_t const ptr1 = hLowPtrOffset + wLowPtrOffset + basePtr;
        v1 = bottomData[ptr1];
    }
    scalar_t v2 = 0;
    if (hLow >= 0 && wHigh <= width - 1)
    {
        int32_t const ptr2 = hLowPtrOffset + wHighPtrOffset + basePtr;
        v2 = bottomData[ptr2];
    }
    scalar_t v3 = 0;
    if (hHigh <= height - 1 && wLow >= 0)
    {
        int32_t const ptr3 = hHighPtrOffset + wLowPtrOffset + basePtr;
        v3 = bottomData[ptr3];
    }
    scalar_t v4 = 0;
    if (hHigh <= height - 1 && wHigh <= width - 1)
    {
        int32_t const ptr4 = hHighPtrOffset + wHighPtrOffset + basePtr;
        v4 = bottomData[ptr4];
    }

    scalar_t const w1 = hh * hw, w2 = hh * lw, w3 = lh * hw, w4 = lh * lw;

    scalar_t const val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
    return val;
}

template <typename scalar_t>
__global__ void ms_deformable_im2col_gpu_kernel(int32_t const n, scalar_t const* dataValue,
    int32_t const* dataSpatialShapes, int32_t const* dataLevelStartIndex, scalar_t const* dataSamplingLoc,
    scalar_t const* dataAttnWeight, int32_t const batchSize, int32_t const spatialSize, int32_t const numHeads, int32_t const channels,
    int32_t const numLevels, int32_t const numQuery, int32_t const numPoint, scalar_t* dataCol)
{
    CUDA_KERNEL_LOOP(index, n)
    {
        int32_t _temp = index;
        int32_t const cCol = _temp % channels;
        _temp /= channels;
        int32_t const samplingIndex = _temp;
        int32_t const mCol = _temp % numHeads;
        _temp /= numHeads;
        _temp /= numQuery;
        int32_t const bCol = _temp;

        scalar_t* dataColPtr = dataCol + index;
        int32_t dataWeightPtr = samplingIndex * numLevels * numPoint;
        int32_t dataLocWPtr = dataWeightPtr << 1;
        int32_t const qidStride = numHeads * channels;
        int32_t const dataValuePtrInitOffset = bCol * spatialSize * qidStride;
        scalar_t col = 0;

        for (int32_t lCol = 0; lCol < numLevels; ++lCol)
        {
            int32_t const levelStartId = dataLevelStartIndex[lCol];
            int32_t const spatialHPtr = lCol << 1;
            int32_t const spatialH = dataSpatialShapes[spatialHPtr];
            int32_t const spatialW = dataSpatialShapes[spatialHPtr + 1];
            scalar_t const* dataValuePtr = dataValue + (dataValuePtrInitOffset + levelStartId * qidStride);
            for (int32_t pCol = 0; pCol < numPoint; ++pCol)
            {
                scalar_t const locW = dataSamplingLoc[dataLocWPtr];
                scalar_t const locH = dataSamplingLoc[dataLocWPtr + 1];
                scalar_t const weight = dataAttnWeight[dataWeightPtr];

                scalar_t const hIm = locH * spatialH - 0.5;
                scalar_t const wIm = locW * spatialW - 0.5;

                if (hIm > -1 && wIm > -1 && hIm < spatialH && wIm < spatialW)
                {
                    col += ms_deform_attn_im2col_bilinear(
                               dataValuePtr, spatialH, spatialW, numHeads, channels, hIm, wIm, mCol, cCol)
                        * weight;
                }

                dataWeightPtr += 1;
                dataLocWPtr += 2;
            }
        }
        *dataColPtr = col;
    }
}


int main() {
    using namespace debug;
    typedef float scalar_t;
    int32_t const batchSize = 20;
    int32_t const spatialSize = 1025;
    int32_t const numHeads = 6;
    int32_t const channels = 3;
    int32_t const numLevels = 2;
    int32_t const numQuery = 30;
    int32_t const numPoint = 3;
    const int32_t DATA_ARRAY_SIZE = batchSize*spatialSize*numHeads*channels;
    const int32_t LOC_ARRAY_SIZE = batchSize*numQuery*numHeads*numLevels*numPoint*2;

    scalar_t dataValue[DATA_ARRAY_SIZE];
    int32_t dataSpatialShapes[numLevels][2]={{25, 25}, {20, 20}};
    int32_t dataLevelStartIndex[numLevels]={0, 625};
    scalar_t dataSamplingLoc[LOC_ARRAY_SIZE];
    scalar_t dataAttnWeight[LOC_ARRAY_SIZE>>1];
    
    for(int i = 0; i < DATA_ARRAY_SIZE; ++i)
    {
        dataValue[i] = 100.f;
    }
    for(int i=0; i<LOC_ARRAY_SIZE; ++i)
    {
        dataSamplingLoc[i] = 0.5f + (float)random() / RAND_MAX;
        printf("%f ",dataSamplingLoc[i]);
    }
    for(int i=0; i<LOC_ARRAY_SIZE/2; ++i)
    {
        dataAttnWeight[i] = 0.5f + (float)random() / RAND_MAX;
    }
    
    scalar_t h_dataCol[batchSize * numQuery * numHeads * channels];

    scalar_t* d_dataValue;
    scalar_t* d_dataSamplingLoc;
    scalar_t* d_dataAttnWeight;
    scalar_t* d_dataCol;
    int32_t* d_dataSpatialShapes;
    int32_t* d_dataLevelStartIndex;

    hipMalloc((void**) &d_dataValue, DATA_ARRAY_SIZE*sizeof(scalar_t));
    hipMalloc((void**) &d_dataSamplingLoc, LOC_ARRAY_SIZE*sizeof(scalar_t));
    hipMalloc((void**) &d_dataAttnWeight, LOC_ARRAY_SIZE/2*sizeof(scalar_t));
    hipMalloc((void**) &d_dataSpatialShapes, numLevels*2*sizeof(int32_t));
    hipMalloc((void**) &d_dataLevelStartIndex, numLevels*sizeof(int32_t));

    hipMalloc((void**) &d_dataCol, batchSize*numQuery*numHeads*channels*sizeof(scalar_t));

    hipMemcpy(d_dataValue, dataValue, DATA_ARRAY_SIZE*sizeof(scalar_t), hipMemcpyHostToDevice);
    hipMemcpy(d_dataSamplingLoc, dataSamplingLoc, LOC_ARRAY_SIZE*sizeof(scalar_t), hipMemcpyHostToDevice);
    hipMemcpy(d_dataAttnWeight, dataAttnWeight, LOC_ARRAY_SIZE/2*sizeof(scalar_t), hipMemcpyHostToDevice);
    hipMemcpy(d_dataSpatialShapes, dataSpatialShapes, numLevels*2*sizeof(int32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_dataLevelStartIndex, dataLevelStartIndex, numLevels*sizeof(int32_t), hipMemcpyHostToDevice);
    
    int32_t const numKernels = batchSize * numQuery * numHeads * channels;
    int32_t const numActualKernels = batchSize * numQuery * numHeads * channels;
    int32_t const numThreads = CUDA_NUM_THREADS;

    std::cout << "The numKernel size is: " << numKernels << std::endl;

    ms_deformable_im2col_gpu_kernel<scalar_t><<<GET_BLOCKS(numActualKernels, numThreads), numThreads, 0>>>(
        numKernels, d_dataValue, d_dataSpatialShapes, d_dataLevelStartIndex, d_dataSamplingLoc, d_dataAttnWeight, batchSize,
        spatialSize, numHeads, channels, numLevels, numQuery, numPoint, d_dataCol);
    
    hipMemcpy(h_dataCol, d_dataCol, numKernels*sizeof(scalar_t), hipMemcpyDeviceToHost);
    for(int i=0;i<numKernels;i++){
        printf("%f ",h_dataCol[i]);
        // printf(((i%4) != 3) ? "\t" : "\n");
      }
    hipFree(d_dataValue);
    hipFree(d_dataSamplingLoc);
    hipFree(d_dataAttnWeight);
    hipFree(d_dataSpatialShapes);
    hipFree(d_dataLevelStartIndex);
    hipFree(d_dataCol);
    checkCudaErrors(hipGetLastError());
    std::cout << "Execution success" << std::endl;
    return 0;
}
